#include "hip/hip_runtime.h"
#include "config.cpp"

#include "kernel_3x3.cu"
#include "kernel_5x5.cu"

struct CliOptions {
    string kernel;
    string input_file;
    string output_file;
    string input_dir;
    string output_dir;
};

const map<string, pair<char, float* > > kernels {
    {
        "box-blur",
        {
            3,
            new float[9] {
                1.0/9, 1.0/9, 1.0/9,
                1.0/9, 1.0/9, 1.0/9,
                1.0/9, 1.0/9, 1.0/9
            }
        }
    },
    {
        "gaussian-blur-3",
        {
            3,
            new float[9] {
                1.0/16, 2.0/16, 1.0/16,
                2.0/16, 4.0/16, 2.0/16,
                1.0/16, 2.0/16, 1.0/16
            }
        }
    },
    {
        "gaussian-blur-5",
        {
            5,
            new float[25] {
                1.0/256, 4.0/256,   6.0/256,  4.0/256, 1.0/256,
                4.0/256, 16.0/256, 24.0/256, 16.0/256, 4.0/256,
                6.0/256, 24.0/256, 36.0/256, 24.0/256, 6.0/256,
                4.0/256, 16.0/256, 24.0/256, 16.0/256, 4.0/256,
                1.0/256, 4.0/256,   6.0/256,  4.0/256, 1.0/256
            }
        }
    },
    {
        "edge-detect",
        {
            3,
            new float[9] {
                -1, -1, -1,
                -1,  8, -1,
                -1, -1, -1
            }
        }
    },
    {
        "emboss",
        {
            3,
            new float[9] {
                -2, -1, 0,
                -1,  1, 1,
                 0,  1, 2
            }
        }
    }
};

int maxBlockSize;
int maxBlockDimX;
int maxBlockDimY;
int maxGridDimX;
int maxGridDimY;

vector<string> get_files_from_dir(string name) {
    DIR* dir;
    struct dirent* entry;
    vector<string> files;

    if (!(dir = opendir(name.c_str())))
        return files;

    while ((entry = readdir(dir)) != NULL) {
        if (strcmp(entry->d_name, ".") && strcmp(entry->d_name, "..")) {
            files.push_back(entry->d_name);
        }
    }
    closedir(dir);

    return files;
}

void getError(hipError_t err) {
    if (err != hipSuccess) {
        printf("CUDA error - %s\n", hipGetErrorString(err));
    }
}

pair<float, float> apply_kernel(unsigned char* input_image, unsigned char* output_image, int width, int height, string kernel_type) {
    unsigned char* host_input_r = new unsigned char[width * height];
    unsigned char* host_input_g = new unsigned char[width * height];
    unsigned char* host_input_b = new unsigned char[width * height];
    unsigned char* host_output_r = new unsigned char[width * height];
    unsigned char* host_output_g = new unsigned char[width * height];
    unsigned char* host_output_b = new unsigned char[width * height];
    
    unsigned char* dev_input_r;
    unsigned char* dev_input_g;
    unsigned char* dev_input_b;
    unsigned char* dev_output_r;
    unsigned char* dev_output_g;
    unsigned char* dev_output_b;

    for (int i = 0; i < 3 * width * height; i += 3) {
        host_input_r[i / 3] = input_image[i];
        host_input_g[i / 3] = input_image[i + 1];
        host_input_b[i / 3] = input_image[i + 2];
    }

    hipStream_t stream;
    hipStreamCreate(&stream);

    float* dev_kernel;

    float ms_outer = 0;
    float ms_inner = 0;
    hipEvent_t start_outer;
    hipEvent_t stop_outer;
    hipEvent_t start_inner;
    hipEvent_t stop_inner;
    hipEventCreate(&start_outer);
    hipEventCreate(&stop_outer);
    hipEventCreate(&start_inner);
    hipEventCreate(&stop_inner);

    hipEventRecord(start_outer);
    hipEventSynchronize(start_outer);

    getError(hipMalloc((void **)&dev_kernel, sqr(kernels.at(kernel_type).first) * sizeof(float)));
    getError(hipMalloc((void **)&dev_input_r, width * height * sizeof(unsigned char)));
    getError(hipMalloc((void **)&dev_input_b, width * height * sizeof(unsigned char)));
    getError(hipMalloc((void **)&dev_input_g, width * height * sizeof(unsigned char)));
    getError(hipMalloc((void **)&dev_output_r, width * height * sizeof(unsigned char)));
    getError(hipMalloc((void **)&dev_output_g, width * height * sizeof(unsigned char)));
    getError(hipMalloc((void **)&dev_output_b, width * height * sizeof(unsigned char)));

    getError(hipMemcpyAsync(dev_kernel, kernels.at(kernel_type).second, sqr(kernels.at(kernel_type).first) * sizeof(float), hipMemcpyHostToDevice, stream));
    getError(hipMemcpyAsync(dev_input_r, host_input_r, width * height * sizeof(unsigned char), hipMemcpyHostToDevice, stream));
    getError(hipMemcpyAsync(dev_input_g, host_input_g, width * height * sizeof(unsigned char), hipMemcpyHostToDevice, stream));
    getError(hipMemcpyAsync(dev_input_b, host_input_b, width * height * sizeof(unsigned char), hipMemcpyHostToDevice, stream));

    int blockDim = BLOCK_DIM;
    int gridDimX = ceil(1.0 * width / blockDim);
    int gridDimY = ceil(1.0 * height / blockDim);

    if (gridDimX > maxGridDimX || gridDimY > maxGridDimY) {
        throw runtime_error("Too big image");
    }

    printf("Device params: block size %d, grid x-dim %d, grid y-dim %d\n", blockDim, gridDimX, gridDimY);

    dim3 blockDims(blockDim, blockDim, 1);
    dim3 gridDims(gridDimX, gridDimY, 1);

    hipEventRecord(start_inner);
    hipEventSynchronize(start_inner);

    if (kernels.at(kernel_type).first == 3) {
        apply_kernel_device_3x3<<<gridDims, blockDims, 0, stream>>>(
            dev_input_r, dev_input_g, dev_input_b,
            dev_output_r, dev_output_g, dev_output_b,
            width, height,
            dev_kernel
        );
    } else if (kernels.at(kernel_type).first == 5) {
        apply_kernel_device_5x5<<<gridDims, blockDims, 0, stream>>>(
            dev_input_r, dev_input_g, dev_input_b,
            dev_output_r, dev_output_g, dev_output_b,
            width, height,
            dev_kernel
        );
    }

    hipEventRecord(stop_inner);
    hipEventSynchronize(stop_inner);
    hipEventElapsedTime(&ms_inner, start_inner, stop_inner);

    getError(hipMemcpyAsync(host_output_r, dev_output_r, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost, stream));
    getError(hipMemcpyAsync(host_output_g, dev_output_g, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost, stream));
    getError(hipMemcpyAsync(host_output_b, dev_output_b, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost, stream));

    getError(hipFree(dev_input_r));
    getError(hipFree(dev_input_g));
    getError(hipFree(dev_input_b));
    getError(hipFree(dev_output_r));
    getError(hipFree(dev_output_g));
    getError(hipFree(dev_output_b));
    getError(hipFree(dev_kernel));

    hipEventRecord(stop_outer);
    hipEventSynchronize(stop_outer);
    hipEventElapsedTime(&ms_outer, start_outer, stop_outer);

    hipStreamDestroy(stream);

    printf("GPU calculation time: %g ms\n", ms_inner);
    printf("GPU calculation + transport time: %g ms\n", ms_outer);

    for (int i = 0; i < 3 * width * height; i += 3) {
        output_image[i] = host_output_r[i / 3];
        output_image[i + 1] = host_output_g[i / 3];
        output_image[i + 2] = host_output_b[i / 3];
    }

    return make_pair(ms_inner, ms_outer);
}

void loadCudaSettings() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    maxBlockSize = prop.maxThreadsPerBlock;
    maxBlockDimX = prop.maxThreadsDim[0];
    maxBlockDimY = prop.maxThreadsDim[1];
    maxGridDimX = prop.maxGridSize[0];
    maxGridDimY = prop.maxGridSize[1];

    printf("CUDA block max size - %d\n", prop.maxThreadsPerBlock);
    printf("CUDA block max dimensions - %d, %d, %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("CUDA grid max dimensions - %d, %d, %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
}

CliOptions parseCli(int argc, char** argv) {
    enum CliCommandState {
        KernelType,
        ImageType,
        BigImageParse,
        SmallImagesParseItems,
        Done,
        Error
    };

    CliCommandState state = KernelType;

    CliOptions options;

    int i = 1;
    while (state != Done) {
        if (i > argc - 1) {
            state = Error;    
        }
        switch (state) {
            case KernelType: {
                bool found = false;
                for (auto const entry: kernels) {
                    if (entry.first == argv[i]) {
                        found = true;
                        options.kernel = entry.first;
                    }
                }
                if (!found) {
                    printf("Error: kernel with given name doesn't exist\n");
                    state = Error;
                }
                state = ImageType;
                i++;
                break;
            }
            case ImageType: {
                if (strcmp(argv[i], "--b") == 0) {
                    state = BigImageParse;
                } else if (strcmp(argv[i], "--s") == 0) {
                    state = SmallImagesParseItems;
                } else {
                    printf("Error: expected flag --b or --s\n");
                    state = Error;
                }
                i++;
                break;
            }
            case BigImageParse: {
                options.input_file = argv[i];
                options.output_file = argv[i + 1];
                i += 2;
                state = Done;
                break;
            }
            case SmallImagesParseItems: {
                options.input_dir = argv[i];
                options.output_dir = argv[i + 1];
                i += 2;
                state = Done;
                break;
            }
            case Error:
                printf("Usage - command <kernel> [--b <input file> <output file> | --s <input dir> <output dir> ]\n");
                printf("Available kernels - ");
                for (auto const entry: kernels) {
                    printf("%s ", entry.first.c_str());
                }
                printf("\n");
                exit(1);
            default:
                throw runtime_error("Unknown cli state");
        }
    }

    return options;
}

pair<float, float> processImage(const char* input_file, const char* output_file, string kernel_type) {
    vector<unsigned char> input_image_raw;
    unsigned int width, height;

    unsigned error = lodepng::decode(input_image_raw, width, height, input_file);
    if (error) {
        cout << "decoder error " << error << ": " << lodepng_error_text(error) << endl;
    }

    unsigned char *input_image = new unsigned char[(input_image_raw.size() * 3) / 4];
    unsigned char *output_image = new unsigned char[(input_image_raw.size() * 3) / 4];
    int inp_iterator = 0;
    for (int i = 0; i < input_image_raw.size(); ++i) {
        if ((i + 1) % 4 != 0) {
            input_image[inp_iterator] = input_image_raw.at(i);
            output_image[inp_iterator] = 255;
            inp_iterator++;
        }
    }

    printf("Image size - %dx%d\n", width, height);

    pair<float, float> time_recording = apply_kernel(input_image, output_image, width, height, kernel_type);

    hipDeviceSynchronize();

    int out_iterator = 0;
    vector<unsigned char> output_image_raw(input_image_raw.size());
    for (int i = 0; i < width * height * 3; ++i) {
        output_image_raw[out_iterator] = output_image[i];
        out_iterator++;
        if ((i + 1) % 3 == 0) {
            output_image_raw[out_iterator] = 255;
            out_iterator++;
        }
    }

    error = lodepng::encode(output_file, output_image_raw, width, height);

    if (error) {
        printf("Encoder error: %s\n", lodepng_error_text(error));
    }

    delete[] input_image;
    delete[] output_image;

    return time_recording;
}

int main(int argc, char** argv) {
    CliOptions options = parseCli(argc, argv);

    loadCudaSettings();

    float total_outer_time = 0;
    float total_inner_time = 0;

    vector<pair<string, string> > images;

    if (options.input_dir != "") {
        for (auto entry : get_files_from_dir(options.input_dir)) {
            cout << entry << endl;
            images.push_back(make_pair(options.input_dir + "/" + entry, options.output_dir + "/" + entry));
        }
    } else {
        images.push_back(make_pair(options.input_file, options.output_file));
    }

    for (int i = 0; i < images.size(); i++) {
        printf("Started processing image %s\n", images[i].first.c_str());

        pair<float, float> time_recording = processImage(images[i].first.c_str(), images[i].second.c_str(), options.kernel);
        total_inner_time += time_recording.first;
        total_outer_time += time_recording.second;

        printf("Finised. Output was written to %s\n", images[i].second.c_str());
    }

    printf("Total time: calculations - %g, calculations + transport - %g\n", total_inner_time, total_outer_time);
    
    return 0;
}