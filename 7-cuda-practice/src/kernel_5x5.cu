#include "hip/hip_runtime.h"
#include "config.cpp"

__global__ void apply_kernel_device_5x5(
    unsigned char* input_image_r,
    unsigned char* input_image_g,
    unsigned char* input_image_b,
    unsigned char* output_image_r,
    unsigned char* output_image_g,
    unsigned char* output_image_b,
    int width,
    int height,
    float* kernel
) {
    const unsigned int linearX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int linearY = blockIdx.y * blockDim.y + threadIdx.y;

    if (linearX >= width || linearY >= height) {
        return;
    }

    bool isInner = (linearX > 1 && linearX < width - 2 && linearY > 2 && linearY < height - 2);

    if (!isInner) {
        output_image_r[linearY * width + linearX] = input_image_r[linearY * width + linearX];
        output_image_g[linearY * width + linearX] = input_image_g[linearY * width + linearX];
        output_image_b[linearY * width + linearX] = input_image_b[linearY * width + linearX];

        return;
    }

    float r = 0;
    float g = 0;
    float b = 0;

    // i = -2
    
    r += input_image_r[(linearY - 2) * width + (linearX - 2)] * kernel[0];
    g += input_image_g[(linearY - 2) * width + (linearX - 2)] * kernel[0];
    b += input_image_b[(linearY - 2) * width + (linearX - 2)] * kernel[0];

    r += input_image_r[(linearY - 2) * width + (linearX - 1)] * kernel[1];
    g += input_image_g[(linearY - 2) * width + (linearX - 1)] * kernel[1];
    b += input_image_b[(linearY - 2) * width + (linearX - 1)] * kernel[1];

    r += input_image_r[(linearY - 2) * width + (linearX)] * kernel[2];
    g += input_image_g[(linearY - 2) * width + (linearX)] * kernel[2];
    b += input_image_b[(linearY - 2) * width + (linearX)] * kernel[2];

    r += input_image_r[(linearY - 2) * width + (linearX + 1)] * kernel[3];
    g += input_image_g[(linearY - 2) * width + (linearX + 1)] * kernel[3];
    b += input_image_b[(linearY - 2) * width + (linearX + 1)] * kernel[3];

    r += input_image_r[(linearY - 2) * width + (linearX + 2)] * kernel[4];
    g += input_image_g[(linearY - 2) * width + (linearX + 2)] * kernel[4];
    b += input_image_b[(linearY - 2) * width + (linearX + 2)] * kernel[4];

    // i = -1
    
    r += input_image_r[(linearY - 1) * width + (linearX - 2)] * kernel[5];
    g += input_image_g[(linearY - 1) * width + (linearX - 2)] * kernel[5];
    b += input_image_b[(linearY - 1) * width + (linearX - 2)] * kernel[5];

    r += input_image_r[(linearY - 1) * width + (linearX - 1)] * kernel[6];
    g += input_image_g[(linearY - 1) * width + (linearX - 1)] * kernel[6];
    b += input_image_b[(linearY - 1) * width + (linearX - 1)] * kernel[6];

    r += input_image_r[(linearY - 1) * width + (linearX)] * kernel[7];
    g += input_image_g[(linearY - 1) * width + (linearX)] * kernel[7];
    b += input_image_b[(linearY - 1) * width + (linearX)] * kernel[7];

    r += input_image_r[(linearY - 1) * width + (linearX + 1)] * kernel[8];
    g += input_image_g[(linearY - 1) * width + (linearX + 1)] * kernel[8];
    b += input_image_b[(linearY - 1) * width + (linearX + 1)] * kernel[8];

    r += input_image_r[(linearY - 1) * width + (linearX + 2)] * kernel[9];
    g += input_image_g[(linearY - 1) * width + (linearX + 2)] * kernel[9];
    b += input_image_b[(linearY - 1) * width + (linearX + 2)] * kernel[9];

    // i = 0
    
    r += input_image_r[(linearY) * width + (linearX - 2)] * kernel[10];
    g += input_image_g[(linearY) * width + (linearX - 2)] * kernel[10];
    b += input_image_b[(linearY) * width + (linearX - 2)] * kernel[10];

    r += input_image_r[(linearY) * width + (linearX - 1)] * kernel[11];
    g += input_image_g[(linearY) * width + (linearX - 1)] * kernel[11];
    b += input_image_b[(linearY) * width + (linearX - 1)] * kernel[11];

    r += input_image_r[(linearY) * width + (linearX)] * kernel[12];
    g += input_image_g[(linearY) * width + (linearX)] * kernel[12];
    b += input_image_b[(linearY) * width + (linearX)] * kernel[12];

    r += input_image_r[(linearY) * width + (linearX + 1)] * kernel[13];
    g += input_image_g[(linearY) * width + (linearX + 1)] * kernel[13];
    b += input_image_b[(linearY) * width + (linearX + 1)] * kernel[13];

    r += input_image_r[(linearY) * width + (linearX + 2)] * kernel[14];
    g += input_image_g[(linearY) * width + (linearX + 2)] * kernel[14];
    b += input_image_b[(linearY) * width + (linearX + 2)] * kernel[14];

    // i = 1
    
    r += input_image_r[(linearY + 1) * width + (linearX - 2)] * kernel[15];
    g += input_image_g[(linearY + 1) * width + (linearX - 2)] * kernel[15];
    b += input_image_b[(linearY + 1) * width + (linearX - 2)] * kernel[15];

    r += input_image_r[(linearY + 1) * width + (linearX - 1)] * kernel[16];
    g += input_image_g[(linearY + 1) * width + (linearX - 1)] * kernel[16];
    b += input_image_b[(linearY + 1) * width + (linearX - 1)] * kernel[16];

    r += input_image_r[(linearY + 1) * width + (linearX)] * kernel[17];
    g += input_image_g[(linearY + 1) * width + (linearX)] * kernel[17];
    b += input_image_b[(linearY + 1) * width + (linearX)] * kernel[17];

    r += input_image_r[(linearY + 1) * width + (linearX + 1)] * kernel[18];
    g += input_image_g[(linearY + 1) * width + (linearX + 1)] * kernel[18];
    b += input_image_b[(linearY + 1) * width + (linearX + 1)] * kernel[18];

    r += input_image_r[(linearY + 1) * width + (linearX + 2)] * kernel[19];
    g += input_image_g[(linearY + 1) * width + (linearX + 2)] * kernel[19];
    b += input_image_b[(linearY + 1) * width + (linearX + 2)] * kernel[19];

    // i = 2
    
    r += input_image_r[(linearY + 2) * width + (linearX - 2)] * kernel[20];
    g += input_image_g[(linearY + 2) * width + (linearX - 2)] * kernel[20];
    b += input_image_b[(linearY + 2) * width + (linearX - 2)] * kernel[20];

    r += input_image_r[(linearY + 2) * width + (linearX - 1)] * kernel[21];
    g += input_image_g[(linearY + 2) * width + (linearX - 1)] * kernel[21];
    b += input_image_b[(linearY + 2) * width + (linearX - 1)] * kernel[21];

    r += input_image_r[(linearY + 2) * width + (linearX)] * kernel[22];
    g += input_image_g[(linearY + 2) * width + (linearX)] * kernel[22];
    b += input_image_b[(linearY + 2) * width + (linearX)] * kernel[22];

    r += input_image_r[(linearY + 2) * width + (linearX + 1)] * kernel[23];
    g += input_image_g[(linearY + 2) * width + (linearX + 1)] * kernel[23];
    b += input_image_b[(linearY + 2) * width + (linearX + 1)] * kernel[23];

    r += input_image_r[(linearY + 2) * width + (linearX + 2)] * kernel[24];
    g += input_image_g[(linearY + 2) * width + (linearX + 2)] * kernel[24];
    b += input_image_b[(linearY + 2) * width + (linearX + 2)] * kernel[24];

    output_image_r[linearY * width + linearX] = ceil(r);
    output_image_g[linearY * width + linearX] = ceil(g);
    output_image_b[linearY * width + linearX] = ceil(b);
}