#include "hip/hip_runtime.h"
#include "config.cpp"

__global__ void apply_kernel_device_3x3(
    unsigned char* input_image_r,
    unsigned char* input_image_g,
    unsigned char* input_image_b,
    unsigned char* output_image_r,
    unsigned char* output_image_g,
    unsigned char* output_image_b,
    int width,
    int height,
    float* kernel
) {
    const unsigned int linearX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int linearY = blockIdx.y * blockDim.y + threadIdx.y;

    if (linearX >= width || linearY >= height) {
        return;
    }

    uint x = threadIdx.x + 1;
    uint y = threadIdx.y + 1;

    __shared__ unsigned char shared_image_r[BLOCK_DIM + 2][BLOCK_DIM + 2];
    __shared__ unsigned char shared_image_g[BLOCK_DIM + 2][BLOCK_DIM + 2];
    __shared__ unsigned char shared_image_b[BLOCK_DIM + 2][BLOCK_DIM + 2];

    shared_image_r[x][y] = input_image_r[linearY * width + linearX];
    shared_image_g[x][y] = input_image_g[linearY * width + linearX];
    shared_image_b[x][y] = input_image_b[linearY * width + linearX];

    bool isInner = (linearX > 0 && linearX < width - 1 && linearY > 0 && linearY < height - 1);

    if (!isInner) {
        output_image_r[linearY * width + linearX] = shared_image_r[x][y];
        output_image_g[linearY * width + linearX] = shared_image_g[x][y];
        output_image_b[linearY * width + linearX] = shared_image_b[x][y];

        return;
    }

    bool isLeftEdge = (threadIdx.x == 0);
    bool isRightEdge = (threadIdx.x == blockDim.x - 1);
    bool isTopEdge = (threadIdx.y == 0);
    bool isBottomEdge = (threadIdx.y == blockDim.y - 1);

    if (isLeftEdge) {
        shared_image_r[x - 1][y] = input_image_r[linearY * width + linearX - 1];
        shared_image_g[x - 1][y] = input_image_g[linearY * width + linearX - 1];
        shared_image_b[x - 1][y] = input_image_b[linearY * width + linearX - 1];
    }

    if (isRightEdge) {
        shared_image_r[x + 1][y] = input_image_r[linearY * width + linearX + 1];
        shared_image_g[x + 1][y] = input_image_g[linearY * width + linearX + 1];
        shared_image_b[x + 1][y] = input_image_b[linearY * width + linearX + 1];
    }

    if (isTopEdge) {
        shared_image_r[x][y - 1] = input_image_r[(linearY - 1) * width + linearX];
        shared_image_g[x][y - 1] = input_image_g[(linearY - 1) * width + linearX];
        shared_image_b[x][y - 1] = input_image_b[(linearY - 1) * width + linearX];
    }

    if (isBottomEdge) {
        shared_image_r[x][y + 1] = input_image_r[(linearY + 1) * width + linearX];
        shared_image_g[x][y + 1] = input_image_g[(linearY + 1) * width + linearX];
        shared_image_b[x][y + 1] = input_image_b[(linearY + 1) * width + linearX];
    }

    if (isTopEdge && isLeftEdge) {
        shared_image_r[x - 1][y - 1] = input_image_r[(linearY - 1) * width + linearX - 1];
        shared_image_g[x - 1][y - 1] = input_image_g[(linearY - 1) * width + linearX - 1];
        shared_image_b[x - 1][y - 1] = input_image_b[(linearY - 1) * width + linearX - 1];
    }

    if (isTopEdge && isRightEdge) {
        shared_image_r[x + 1][y - 1] = input_image_r[(linearY - 1) * width + linearX + 1];
        shared_image_g[x + 1][y - 1] = input_image_g[(linearY - 1) * width + linearX + 1];
        shared_image_b[x + 1][y - 1] = input_image_b[(linearY - 1) * width + linearX + 1];
    }

    if (isBottomEdge && isLeftEdge) {
        shared_image_r[x - 1][y + 1] = input_image_r[(linearY + 1) * width + linearX - 1];
        shared_image_g[x - 1][y + 1] = input_image_g[(linearY + 1) * width + linearX - 1];
        shared_image_b[x - 1][y + 1] = input_image_b[(linearY + 1) * width + linearX - 1];
    }

    if (isBottomEdge && isRightEdge) {
        shared_image_r[x + 1][y + 1] = input_image_r[(linearY + 1) * width + linearX + 1];
        shared_image_g[x + 1][y + 1] = input_image_g[(linearY + 1) * width + linearX + 1];
        shared_image_b[x + 1][y + 1] = input_image_b[(linearY + 1) * width + linearX + 1];
    }

    __syncthreads();

    float r = 0;
    float g = 0;
    float b = 0;

    // i = -1

    r += shared_image_r[x - 1][y - 1] * kernel[0];
    g += shared_image_g[x - 1][y - 1] * kernel[0];
    b += shared_image_b[x - 1][y - 1] * kernel[0];

    r += shared_image_r[x - 1][y] * kernel[1];
    g += shared_image_g[x - 1][y] * kernel[1];
    b += shared_image_b[x - 1][y] * kernel[1];

    r += shared_image_r[x - 1][y + 1] * kernel[2];
    g += shared_image_g[x - 1][y + 1] * kernel[2];
    b += shared_image_b[x - 1][y + 1] * kernel[2];

    // i = 0

    r += shared_image_r[x][y - 1] * kernel[3];
    g += shared_image_g[x][y - 1] * kernel[3];
    b += shared_image_b[x][y - 1] * kernel[3];

    r += shared_image_r[x][y] * kernel[4];
    g += shared_image_g[x][y] * kernel[4];
    b += shared_image_b[x][y] * kernel[4];

    r += shared_image_r[x][y + 1] * kernel[5];
    g += shared_image_g[x][y + 1] * kernel[5];
    b += shared_image_b[x][y + 1] * kernel[5];

    // i = 1

    r += shared_image_r[x + 1][y - 1] * kernel[6];
    g += shared_image_g[x + 1][y - 1] * kernel[6];
    b += shared_image_b[x + 1][y - 1] * kernel[6];

    r += shared_image_r[x + 1][y] * kernel[7];
    g += shared_image_g[x + 1][y] * kernel[7];
    b += shared_image_b[x + 1][y] * kernel[7];

    r += shared_image_r[x + 1][y + 1] * kernel[8];
    g += shared_image_g[x + 1][y + 1] * kernel[8];
    b += shared_image_b[x + 1][y + 1] * kernel[8];

    output_image_r[linearY * width + linearX] = ceil(r);
    output_image_g[linearY * width + linearX] = ceil(g);
    output_image_b[linearY * width + linearX] = ceil(b);
}